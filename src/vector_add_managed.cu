#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handling arbitrary vector size
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}

int main(){
    // Allocate managed memory
    float *x, *y, *z;
    hipMallocManaged((void**)&x, sizeof(float) * N);
    hipMallocManaged((void**)&y, sizeof(float) * N);
    hipMallocManaged((void**)&z, sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Executing kernel 
    int block_size = 256;
    int grid_size = ((N + block_size - 1) / block_size);
    vector_add<<<grid_size,block_size>>>(z, x, y, N);
    
    // 同步 Device 保证结果能正确访问
    hipDeviceSynchronize();
  
    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(z[i] - x[i] - y[i]) < MAX_ERR);
    }

    printf("PASSED\n");

    // Deallocate managed memory
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}

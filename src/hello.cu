
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
    printf( "\"Hello, world!\", says the GPU.\n" );
}

void hello_from_cpu()
{
    printf( "\"Hello, world!\", says the CPU.\n" );
}

// host code entrance
int main( int argc, char **argv )
{
    hello_from_cpu();
    hello_from_gpu <<< 2, 4>>>();
    hipDeviceReset();
    return 0;
}

